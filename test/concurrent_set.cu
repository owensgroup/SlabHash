/*
 * Copyright 2019 Saman Ashkiani
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied. See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <random>
#include <vector>
#include "gpu_hash_table.cuh"
#include "slab_hash.cuh"
//=======================================
#define DEVICE_ID 0

int main(int argc, char** argv) {
  //=========
  int devCount;
  hipGetDeviceCount(&devCount);
  hipDeviceProp_t devProp;
  if (devCount) {
    hipSetDevice(DEVICE_ID);  // be changed later
    hipGetDeviceProperties(&devProp, DEVICE_ID);
  }
  printf("Device: %s\n", devProp.name);

  //======================================
  // Building my hash table:
  //======================================
  uint32_t num_keys = 1<<20;

  float expected_chain = 0.6f;
  uint32_t num_elements_per_unit = 15;
  uint32_t expected_elements_per_bucket =
      expected_chain * num_elements_per_unit;
  uint32_t num_buckets = (num_keys + expected_elements_per_bucket - 1) /
                         expected_elements_per_bucket;

  // ==== generating key-values and queries on the host:
  float existing_ratio = 1.0f;  // ratio of queries within the table
  uint32_t num_queries = num_keys;

  using KeyT = uint32_t;
  using ValueT = uint32_t;
  auto num_elements = 2 * num_keys;

  std::vector<KeyT> h_key(num_elements);
  std::vector<ValueT> h_value(num_elements);
  std::vector<KeyT> h_query(num_queries);
  std::vector<ValueT> h_correct_result(num_queries);
  std::vector<ValueT> h_result(num_queries);

  // std::iota(h_key.begin(), h_key.end(), 0);
  const auto f = [](const KeyT& key) { return key * 10; };

  std::random_device rd;
  const int64_t seed = 1;
  std::mt19937 rng(seed);
  std::vector<uint32_t> index(num_elements);
  std::iota(index.begin(), index.end(), 0);
  std::shuffle(index.begin(), index.end(), rng);

  for (int32_t i = 0; i < index.size(); i++) {
    h_key[i] = index[i];
    h_value[i] = f(h_key[i]);
  }

  //=== generating random queries with a fixed ratio existing in keys
  uint32_t num_existing = static_cast<uint32_t>(existing_ratio * num_queries);

  for (int i = 0; i < num_existing; i++) {
    h_query[i] = h_key[num_keys - 1 - i];
    h_correct_result[i] = f(h_query[i]);
  }

  for (int i = 0; i < (num_queries - num_existing); i++) {
    h_query[num_existing + i] = h_key[num_keys + i];
    h_correct_result[num_existing + i] = SEARCH_NOT_FOUND;
  }
  // permuting the queries:
  std::vector<int> q_index(num_queries);
  std::iota(q_index.begin(), q_index.end(), 0);
  std::shuffle(q_index.begin(), q_index.end(), rng);
  for (int i = 0; i < num_queries; i++) {
    std::swap(h_query[i], h_query[q_index[i]]);
    std::swap(h_correct_result[i], h_correct_result[q_index[i]]);
  }
  // gpu_hash_table<KeyT, ValueT, DEVICE_ID> hash_table(num_keys, num_buckets, seed);
  GpuSlabHash<KeyT, ValueT, DEVICE_ID, SlabHashType::PhaseConcurrentMap> mySet(1, nullptr);

  // float build_time =
  //     hash_table.hash_build(h_key.data(), h_value.data(), num_keys);
  // float search_time =
  //     hash_table.hash_search(h_query.data(), h_result.data(), num_queries);
  // float search_time_bulk =
  //     hash_table.hash_search_bulk(h_query.data(), h_result.data(), num_queries);
  // // // hash_table.print_bucket(0);
  // printf("Hash table: \n");
  // printf("num_keys = %d, num_buckets = %d\n", num_keys, num_buckets);
  // // printf("\t1) Hash table init in %.3f ms\n", init_time);
  // printf("\t2) Hash table built in %.3f ms (%.3f M elements/s)\n", build_time,
  //        double(num_keys) / build_time / 1000.0);
  // printf("\t3) Hash table search (%.2f) in %.3f ms (%.3f M queries/s)\n",
  //        existing_ratio, search_time,
  //        double(num_queries) / search_time / 1000.0);
  // printf("\t4) Hash table bulk search (%.2f) in %.3f ms (%.3f Mqueries/s)\n",
  //        existing_ratio, search_time_bulk,
  //        double(num_queries) / search_time_bulk / 1000.0);

  // double load_factor = hash_table.measureLoadFactor();

  // printf("The load factor is %.2f, number of buckets %d\n", load_factor,
  //        num_buckets);

  // // ==== validation:
  // for (int i = 0; i < num_queries; i++) {
  //   if (h_correct_result[i] != h_result[i]) {
  //     printf("### wrong result at index %d: [%d] -> %d, but should be %d\n", i,
  //            h_query[i], h_result[i], h_correct_result[i]);
  //     break;
  //   }
  //   if (i == (num_queries - 1))
  //     printf("Validation done successfully\n");
  // }

  // delete[] h_key;
  // delete[] h_value;
  // delete[] h_result;
  // delete[] h_correct_result;
  // delete[] h_query;
  // delete gpu_hash_table_ptr;
}