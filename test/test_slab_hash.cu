/*
 * Copyright 2018 Saman Ashkiani
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied. See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include "gpu_hash_table.cuh"
#include "slab_alloc.cuh"
#include "slab_hash.cuh"
#define DEVICE_ID 0

int main(int argc, char** argv) {
  //=========
  int devCount;
  hipGetDeviceCount(&devCount);
  hipDeviceProp_t devProp;
  if (devCount) {
    hipSetDevice(DEVICE_ID);  // be changed later
    hipGetDeviceProperties(&devProp, DEVICE_ID);
  }
  printf("Device: %s\n", devProp.name);

  auto my_hash_table = new gpu_hash_table<uint32_t, uint32_t, DEVICE_ID>(100, 10, /*seed = */ 1);

  std::vector<uint32_t> h_key {10,5,1};
  std::vector<uint32_t> h_value {100,50,10};

  my_hash_table->hash_build(h_key.data(), h_value.data(), h_key.size());
  // auto slab_alloc = new SlabAllocLight<8, 32, 1>();
  // printf("slab alloc constructed\n");

  // delete slab_alloc;

  // auto slab_hash =
  //     new GpuSlabHash<uint32_t, uint32_t, SlabHashType::ConcurrentMap>();
  // std::cout << slab_hash->to_string() << std::endl;
  delete my_hash_table;
  return 0;
}