/*
 * Copyright 2018 Saman Ashkiani
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
 * implied. See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "slab_alloc.cuh"
#include "slab_hash.cuh"
#define DEVICE_ID 0

int main(int argc, char** argv){
	//=========
	int devCount;
  hipGetDeviceCount(&devCount);
  hipDeviceProp_t devProp;
  if(devCount){
    hipSetDevice(DEVICE_ID); // be changed later
    hipGetDeviceProperties(&devProp, DEVICE_ID);
  }
  printf("Device: %s\n", devProp.name);

  auto slab_alloc = new SlabAllocLight<8,32,1>();
  printf("slab alloc constructed\n");

  delete slab_alloc;

  auto slab_hash = new GpuSlabHash<uint32_t, uint32_t, SlabHashType::ConcurrentMap> ();
  
  return 0;
}